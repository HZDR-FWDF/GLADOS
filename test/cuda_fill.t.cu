#include <algorithm>
#include <cstdlib>

#define BOOST_TEST_MODULE CudaFill
#include <boost/test/unit_test.hpp>

#include <ddrf/cuda/algorithm.h>
#include <ddrf/cuda/memory.h>
#include <ddrf/cuda/sync_policy.h>

BOOST_AUTO_TEST_CASE(cuda_fill_sync_1d)
{
    constexpr auto szx = 4096;
    constexpr auto dim = szx;

    auto host_orig = ddrf::cuda::make_unique_pinned_host<int>(szx);
    auto host_dest = ddrf::cuda::make_unique_pinned_host<int>(szx);
    auto dev = ddrf::cuda::make_unique_device<int>(szx);

    auto ho = host_orig.get();
    auto hd = host_dest.get();

    constexpr auto host_val = 0x01010101;
    constexpr auto dev_val = 0x01;

    std::fill(ho, ho + dim, host_val);
    std::generate(hd, hd + dim, std::rand);

    ddrf::cuda::fill(ddrf::cuda::sync, dev, dev_val, szx);
    ddrf::cuda::copy(ddrf::cuda::sync, host_dest, dev, szx);

    BOOST_CHECK(std::equal(ho, ho + dim, hd));
}

BOOST_AUTO_TEST_CASE(cuda_fill_sync_2d)
{
    constexpr auto szx = 64;
    constexpr auto szy = 64;
    constexpr auto dim = szx * szy;

    auto host_orig = ddrf::cuda::make_unique_pinned_host<int>(szx, szy);
    auto host_dest = ddrf::cuda::make_unique_pinned_host<int>(szx, szy);
    auto dev = ddrf::cuda::make_unique_device<int>(szx, szy);

    auto ho = host_orig.get();
    auto hd = host_dest.get();

    constexpr auto host_val = 0x01010101;
    constexpr auto dev_val = 0x01;

    std::fill(ho, ho + dim, host_val);
    std::generate(hd, hd + dim, std::rand);

    ddrf::cuda::fill(ddrf::cuda::sync, dev, dev_val, szx, szy);
    ddrf::cuda::copy(ddrf::cuda::sync, host_dest, dev, szx, szy);

    BOOST_CHECK(std::equal(ho, ho + dim, hd));
}

BOOST_AUTO_TEST_CASE(cuda_fill_sync_3d)
{
    constexpr auto szx = 8;
    constexpr auto szy = 8;
    constexpr auto szz = 8;
    constexpr auto dim = szx * szy * szz;

    auto host_orig = ddrf::cuda::make_unique_pinned_host<int>(szx, szy, szz);
    auto host_dest = ddrf::cuda::make_unique_pinned_host<int>(szx, szy, szz);
    auto dev = ddrf::cuda::make_unique_device<int>(szx, szy, szz);

    auto ho = host_orig.get();
    auto hd = host_dest.get();

    constexpr auto host_val = 0x01010101;
    constexpr auto dev_val = 0x01;

    std::fill(ho, ho + dim, host_val);
    std::generate(hd, hd + dim, std::rand);

    ddrf::cuda::fill(ddrf::cuda::sync, dev, dev_val, szx, szy, szz);
    ddrf::cuda::copy(ddrf::cuda::sync, host_dest, dev, szx, szy, szz);

    BOOST_CHECK(std::equal(ho, ho + dim, hd));
}

BOOST_AUTO_TEST_CASE(cuda_fill_async_1d)
{
    constexpr auto szx = 4096;
    constexpr auto dim = szx;

    auto host_orig = ddrf::cuda::make_unique_pinned_host<int>(szx);
    auto host_dest = ddrf::cuda::make_unique_pinned_host<int>(szx);
    auto dev = ddrf::cuda::make_unique_device<int>(szx);

    auto ho = host_orig.get();
    auto hd = host_dest.get();

    constexpr auto host_val = 0x01010101;
    constexpr auto dev_val = 0x01;

    std::fill(ho, ho + dim, host_val);
    std::generate(hd, hd + dim, std::rand);

    ddrf::cuda::fill(ddrf::cuda::async, dev, dev_val, szx);
    hipDeviceSynchronize();
    ddrf::cuda::copy(ddrf::cuda::async, host_dest, dev, szx);
    hipDeviceSynchronize();

    BOOST_CHECK(std::equal(ho, ho + dim, hd));
}

BOOST_AUTO_TEST_CASE(cuda_fill_async_2d)
{
    constexpr auto szx = 64;
    constexpr auto szy = 64;
    constexpr auto dim = szx * szy;

    auto host_orig = ddrf::cuda::make_unique_pinned_host<int>(szx, szy);
    auto host_dest = ddrf::cuda::make_unique_pinned_host<int>(szx, szy);
    auto dev = ddrf::cuda::make_unique_device<int>(szx, szy);

    auto ho = host_orig.get();
    auto hd = host_dest.get();

    constexpr auto host_val = 0x01010101;
    constexpr auto dev_val = 0x01;

    std::fill(ho, ho + dim, host_val);
    std::generate(hd, hd + dim, std::rand);

    ddrf::cuda::fill(ddrf::cuda::async, dev, dev_val, szx, szy);
    hipDeviceSynchronize();
    ddrf::cuda::copy(ddrf::cuda::async, host_dest, dev, szx, szy);
    hipDeviceSynchronize();

    BOOST_CHECK(std::equal(ho, ho + dim, hd));
}

BOOST_AUTO_TEST_CASE(cuda_fill_async_3d)
{
    constexpr auto szx = 8;
    constexpr auto szy = 8;
    constexpr auto szz = 8;
    constexpr auto dim = szx * szy * szz;

    auto host_orig = ddrf::cuda::make_unique_pinned_host<int>(szx, szy, szz);
    auto host_dest = ddrf::cuda::make_unique_pinned_host<int>(szx, szy, szz);
    auto dev = ddrf::cuda::make_unique_device<int>(szx, szy, szz);

    auto ho = host_orig.get();
    auto hd = host_dest.get();

    constexpr auto host_val = 0x01010101;
    constexpr auto dev_val = 0x01;

    std::fill(ho, ho + dim, host_val);
    std::generate(hd, hd + dim, std::rand);

    ddrf::cuda::fill(ddrf::cuda::async, dev, dev_val, szx, szy, szz);
    hipDeviceSynchronize();
    ddrf::cuda::copy(ddrf::cuda::async, host_dest, dev, szx, szy, szz);
    hipDeviceSynchronize();

    BOOST_CHECK(std::equal(ho, ho + dim, hd));
}
